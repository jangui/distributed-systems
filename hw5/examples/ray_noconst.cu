#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* 
 * This copy of code is a derivative designed for educational purposes 
 * and it contains source code provided by NVIDIA Corporation.
 *
*/

#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1024

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

struct Sphere {

    float   r,b,g;// color of the sphere
    float   radius; 
    float   x,y,z;// coordinate of the center
    
    // will return the distance between imaginary camera and hit
    __device__ float hit( float ox, float oy, float *n ) {
        float dx = ox - x; // distance on x-axis
        float dy = oy - y; // distance on y-axis
        // if (dx*dx + dy*dy > radius*radius), ray will not hit sphere
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            // n is used in visual effect
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

#define SPHERES 20

__global__ void kernel( Sphere *s, unsigned char *ptr ) {

    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // this is a linear offset into output buffer
    int offset = x + y * blockDim.x * gridDim.x;

    // shift the (x,y) image coordinate so that z-axis go through center
    float   ox = (x - DIM/2);
    float   oy = (y - DIM/2);

    float   r=0, g=0, b=0;// set the background to black
    float   maxz = -INF;
    for(int i=0; i<SPHERES; i++) {
        float   n;
        float   t = s[i].hit( ox, oy, &n ); // return the distance
        if (t > maxz) { 
            float fscale = n;// improve visual effect
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t; // update maxz everytime a smaller distance is found
        }
    } 

    // color the bitmap according to what the ray has 'seen'
    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
    Sphere          *s;
};

int main( void ) {

    // declare the data block and other needed variables
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char   *dev_bitmap;
    Sphere          *s;

    // allocate temp memory for the Sphere dataset on CPU
    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );

    // initialize the Sphere dataset
    for (int i=0; i<SPHERES; i++) {
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 1000.0f ) - 500;
        temp_s[i].y = rnd( 1000.0f ) - 500;
        temp_s[i].z = rnd( 1000.0f ) - 500;
        temp_s[i].radius = rnd( 100.0f ) + 20;
    }

    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // allocate memory on the GPU for the output bitmap
    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );

    // allocate memory for the Sphere dataset on GPU
    HANDLE_ERROR( hipMalloc( (void**)&s, sizeof(Sphere) * SPHERES ) );

    // transfer the initialized Sphere dataset from CPU memory to GPU memory
    HANDLE_ERROR( hipMemcpy( s, temp_s, sizeof(Sphere) * SPHERES,
                                hipMemcpyHostToDevice ) );

    // generate a bitmap from our sphere data
    dim3    grids(DIM/32,DIM/32);
    dim3    threads(32,32);
    kernel<<<grids,threads>>>( s, dev_bitmap );

    // copy our bitmap back from the GPU for display
    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    // free CPU memory
    free( temp_s );

    // free GPU memory
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    HANDLE_ERROR( hipFree( dev_bitmap ) );
    HANDLE_ERROR( hipFree( s ) );

    // display
    bitmap.display_and_exit();
}

