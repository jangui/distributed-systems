#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "hip/hip_runtime.h"

#define blockD 32
   
main(void){

    void MatrixMultiplication(float *, float *, float *, int);

    const int Width = 1024;

    int size = Width * Width * sizeof(float);
    float *M, *N, *P;   
    
    // allocate memory on the CPU
    M = (float*)malloc(size);
    N = (float*)malloc(size);
    P = (float*)malloc(size);

    // initialize the matrices
    for (int y=0; y<Width; y++) {
	    for (int x=0; x<Width; x++){
	   		M[y*Width + x] = x + y*Width;
       		N[y*Width + x] = x + y*Width; 
	   }
    }

    MatrixMultiplication(M, N, P, Width);

    // free the memory allocated on the CPU
    free( M );
    free( N );
    free( P );

    return 0;
}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
  // declare cache in the shared memory
  __shared__ float Mds[blockD][blockD];
  __shared__ float Nds[blockD][blockD];
 
  // keep track of column index of the Pd element using thread index
  int x = threadIdx.x + blockIdx.x * blockDim.x; // x is column
  // keep track of row index of the Pd element using thread index
  int y = threadIdx.y + blockIdx.y * blockDim.y; // y is row

  float Pvalue = 0;
  // Loop over the Md and Nd block dimension required to compute the Pd element
  for (int m = 0; m < Width/blockD; m++){
	
    // collaboratively loading of Md and Nd blocks into shared memory	 
    Mds[threadIdx.y][threadIdx.x] = Md[y * Width + (m * blockD + threadIdx.x)];
    Nds[threadIdx.y][threadIdx.x] = Nd[(m * blockD + threadIdx.y) * Width + x];
    __syncthreads();
    
    // keep track of the running sum    
    for (int k = 0; k < blockD; k++)
      Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
    __syncthreads();
  }
  
  // write back to the global memory
  Pd[y * Width + x] = Pvalue;
}

void MatrixMultiplication(float *M, float *N, float *P, int Width) {

    int size = Width * Width * sizeof(float);
    float *Md, *Nd, *Pd; 

    // capture start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // allocate memory on the GPU
    HANDLE_ERROR( hipMalloc((void**)&Md, size) );
    HANDLE_ERROR( hipMalloc((void**)&Nd, size) );
    HANDLE_ERROR( hipMalloc((void**)&Pd, size) );

    // transfer M and N to device memory
    HANDLE_ERROR( hipMemcpy(Md, M, size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(Nd, N, size, hipMemcpyHostToDevice) );

    // kernel invocation code
    dim3 dimBlock(blockD, blockD);
    dim3 dimGrid(Width/blockD, Width/blockD);
    MatrixMulKernel<<<dimGrid, dimBlock>>>( Md, Nd, Pd, Width);

    // transfer P from device    
    HANDLE_ERROR( hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost) );

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree(Md) );
    HANDLE_ERROR( hipFree(Nd) );
    HANDLE_ERROR( hipFree(Pd) );

    // destroy events to free memory
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
}
