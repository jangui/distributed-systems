#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* 
 * This copy of code is a derivative designed for educational purposes 
 * and it contains source code provided by NVIDIA Corporation.
 *
*/

#include "../common/book.h"

#define imin(a,b) (a<b?a:b)

const int N = 1024 * 1024;
const int numThread = 256;
const int numBlock = imin( 32, (N+numThread-1) / numThread );

__global__ void dot( float *a, float *b, float *c ) {

    // declare cache in the shared memory
    __shared__ float cache[numThread];

    // keep track of thread index
    int tid = threadIdx.x + blockIdx.x * numThread;
    // connect thread index and cache index
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += numThread * numBlock;// increase by the total number of thread in a grid
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, numThread must be a power of 2 because of the following code
    int i = numThread/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    // write back to the global memory
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int main( void ) {
    float   *a, *b, c, *partial_c;
    float   *dev_a, *dev_b, *dev_partial_c;

    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    b = (float*)malloc( N*sizeof(float) );
    partial_c = (float*)malloc( numBlock*sizeof(float) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // start the timer 
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              N*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_partial_c,
                              numBlock*sizeof(float) ) );

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(float),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N*sizeof(float),
                              hipMemcpyHostToDevice ) ); 

    dot<<<numBlock,numThread>>>( dev_a, dev_b,
                                            dev_partial_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( partial_c, dev_partial_c,
                              numBlock*sizeof(float),
                              hipMemcpyDeviceToHost ) );

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<numBlock; i++) {
        c += partial_c[i];
    }

    #define sum_squares(x)  (x*(x+1)*(2*x+1)/6)
    printf( "Does GPU value %.6g = %.6g?\n", c,
             2 * sum_squares( (float)(N - 1) ) );

    // free memory on the gpu side
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_partial_c ) );

    // free memory on the cpu side
    free( a );
    free( b );
    free( partial_c );
}
