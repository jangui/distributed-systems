/*
    Jeff Epstein
    NYU Tandon, CS-UY 3254
    Conway's Life in CUDA
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <X11/Xlib.h>
#include <unistd.h>
#include <time.h>


/***********************
    Data structures
************************/

#define GRID_SIZE 512
#define CELL_SIZE 2
#define DELAY 10000


struct global {
    char *cells;
    char *cells_next;
    int *cellsInt; // used to make CUDA updates simpler
    // CUDA vars
    int *gpu_cells;
    int *gpu_cells_next;
};

#ifdef CUDA

#define BLOCKS 128
#define BLOCK_THREADS 128

/***********************
    Game of Life, GPU version
************************/

static void HandleError( cudaError_t err,
                         const char *file,
                         int line ) {
    if (err != cudaSuccess) {
        printf( "%s in %s at line %d\n", cudaGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ int count_neighbors(int *gpu_cells, uint tid, int grid_size) {
    int x = ((tid / grid_size)+grid_size/2)%grid_size;
    int y = ((tid % grid_size)+grid_size/2)%grid_size;
    int count = 0;
    for (int i=x-1; i<=x+1; i++)
        for (int j=y-1; j<=y+1; j++)
            if (i!=x || j!=y)
                count += gpu_cells[((i+grid_size/2)%grid_size) * grid_size + ((j+grid_size/2)%grid_size)]; 
    return count;
}


__device__ void update_cell(int *gpu_cells, int *gpu_cells_next, uint tid, int grid_size) {
    int neighbors = count_neighbors(gpu_cells, tid, grid_size);
    gpu_cells_next[tid] = neighbors==3 || (gpu_cells[tid] && (neighbors == 2 || neighbors == 3));
}

__global__ void kernel(int *gpu_cells, int *gpu_cells_next, int grid_size) {
    // get thread index
    uint thread_index = threadIdx.x + blockDim.x * blockIdx.x;

    while (thread_index < grid_size*grid_size) {
        // Do computation for this thread index
        update_cell(gpu_cells, gpu_cells_next, thread_index, grid_size);
        // update thread index
        thread_index += blockDim.x * gridDim.x;
    }
}

// copy gpu_cells_next to gpu_cells
// must be run after full update finished
__global__ void copyGPUcells(int *gpu_cells, int *gpu_cells_next, int grid_size) {
    // get thread index
    uint thread_index = threadIdx.x + blockDim.x * blockIdx.x;
    while (thread_index < grid_size*grid_size) {
        gpu_cells[thread_index] = gpu_cells_next[thread_index];
        thread_index += blockDim.x * gridDim.x;
    }
}

void init_global(struct global *g) {
    // Initialize the global data structure
    const int size = GRID_SIZE*GRID_SIZE/8;
    g->cells = (char*)calloc(size, sizeof(char)); // calloc will init mem to 0's
    g->cellsInt = (int*)calloc(GRID_SIZE*GRID_SIZE, sizeof(int));

    if (g->cells==NULL || g->cellsInt==NULL) {
        fprintf(stderr, "Error: alloc failed\n");
        exit(1);
    }

    // allocate space on gpu for cells
    HANDLE_ERROR(cudaMalloc((void**)&(g->gpu_cells), GRID_SIZE*GRID_SIZE * sizeof(int)));
    HANDLE_ERROR(cudaMalloc((void**)&(g->gpu_cells_next), GRID_SIZE*GRID_SIZE * sizeof(int)));

    // set initial state to all 0's
    HANDLE_ERROR(cudaMemcpy(g->gpu_cells, g->cellsInt, GRID_SIZE*GRID_SIZE * sizeof(int), cudaMemcpyHostToDevice));
}

bool get_cell(struct global *g, int x, int y) {
    return (g->cells[(y*GRID_SIZE + x)/8] & (1<<(x%8))) != 0;
}

void set_cell(struct global *g, int x, int y, bool val) {
    if (val)
        g->cells[(y*GRID_SIZE+x)/8] |= (1<<(x%8));
    else
        g->cells[(y*GRID_SIZE+x)/8] &= ~(1<<(x%8));
}

void update(struct global *global) {
    // Conway's life algorithm on the GPU

    // update cells
    kernel<<<BLOCKS, BLOCK_THREADS>>>(global->gpu_cells, global->gpu_cells_next, GRID_SIZE);

    // copy gpu_cells_next to gpu_cells
    copyGPUcells<<<BLOCKS, BLOCK_THREADS>>>(global->gpu_cells, global->gpu_cells_next, GRID_SIZE);

    // check if errors
    cudaError err = cudaGetLastError();
    if (cudaSuccess != err)
        fprintf(stderr, "Error %s\n", cudaGetErrorString(err));

    // copy data from gpu to cpu
    cudaMemcpy(global->cellsInt, global->gpu_cells, GRID_SIZE*GRID_SIZE * sizeof(int), cudaMemcpyDeviceToHost);

    // set cpu cells (char) appropriately from cellsInt
    int lower = -GRID_SIZE/2 + 1;
    int upper = GRID_SIZE/2 - 1;
    for (int x = lower; x < upper; x++) {
        for (int y = lower; y < upper; y++) {
            int index = ((x+GRID_SIZE/2)%GRID_SIZE)*GRID_SIZE+((y+GRID_SIZE/2)%GRID_SIZE);
            set_cell(global,x+GRID_SIZE/2,y+GRID_SIZE/2,global->cellsInt[index]);
        }
    }
}

#else

/***********************
    Game of Life, CPU version
************************/

/*
    Allocate memory for data structures
    and initialize data
*/
void init_global(struct global *g) {
    const int size = GRID_SIZE*GRID_SIZE/8;
    g->cells=(char*)malloc(size);
    g->cells_next=(char*)malloc(size);
    if (g->cells==NULL || g->cells_next==NULL) {
        fprintf(stderr, "Error: can't alloc data\n");
        exit(1);
    }
    for (int i=0; i<size; i++)
        g->cells[i]=0;
}

/*
    Returns true if a cell is alive at the given location
*/
bool get_cell(struct global *g, int x, int y) {
    return (g->cells[(y*GRID_SIZE + x)/8] & (1<<(x%8))) != 0;
}

void set_cell_next(struct global *g, int x, int y, bool val) {
    if (val)
        g->cells_next[(y*GRID_SIZE+x)/8] |= (1<<(x%8));
    else
        g->cells_next[(y*GRID_SIZE+x)/8] &= ~(1<<(x%8));
}

/*
    Set a cell alive or dead at the given location
*/
void set_cell(struct global *g, int x, int y, bool val) {
    if (val)
        g->cells[(y*GRID_SIZE+x)/8] |= (1<<(x%8));
    else
        g->cells[(y*GRID_SIZE+x)/8] &= ~(1<<(x%8));
}

/*
    Count neighbors of given cell
*/
int count_neighbors(struct global *g, int x, int y) {
    int count =0;
    for (int i=x-1; i<=x+1; i++)
        for (int j=y-1; j<=y+1; j++)
            if (i!=x || j!=y)
                count += get_cell(g,i,j); 
    return count;
}

/*
    Perform a complete step, storing the new state
    in global->cells
*/
void update(struct global *global) {
    for (int x=1; x<GRID_SIZE-1; x++)
        for (int y=1; y<GRID_SIZE-1; y++) {
            int neighbors = count_neighbors(global, x, y);
            bool newstate = 
                neighbors==3 || (get_cell(global,x,y) && (neighbors == 2 || neighbors == 3));
            set_cell_next(global,x,y,newstate);
        }    
    char *temp=global->cells;
    global->cells = global->cells_next;
    global->cells_next = temp;
}

#endif

/***********************
    X Window stuff
************************/

#define COLOR_RED "#FF0000"
#define COLOR_GREEN "#00FF00"
#define COLOR_BLACK "#000000"
#define COLOR_WHITE "#FFFFFF"

struct display
{
    Display         *display;
    Window          window;
    int             screen;
    Atom            delete_window;
    GC              gc;
    XColor          color1;
    XColor          color2;
    Colormap        colormap;
};

void init_display(struct display *dpy) {
        dpy->display = XOpenDisplay(NULL);
        if(dpy->display == NULL)
        {
            fprintf(stderr, "Error: could not open X dpy->display\n");
            exit(1);
        }
        dpy->screen = DefaultScreen(dpy->display);
        dpy->window = XCreateSimpleWindow(dpy->display, RootWindow(dpy->display, dpy->screen),
                0, 0, GRID_SIZE * CELL_SIZE, 
                GRID_SIZE * CELL_SIZE, 1,
                BlackPixel(dpy->display, dpy->screen), WhitePixel(dpy->display, dpy->screen));
        dpy->delete_window = XInternAtom(dpy->display, "WM_DELETE_WINDOW", 0);
        XSetWMProtocols(dpy->display, dpy->window, &dpy->delete_window, 1);
        XSelectInput(dpy->display, dpy->window, ExposureMask | KeyPressMask);
        XMapWindow(dpy->display, dpy->window);
        dpy->colormap = DefaultColormap(dpy->display, 0);
        dpy->gc = XCreateGC(dpy->display, dpy->window, 0, 0);
        XParseColor(dpy->display, dpy->colormap, COLOR_BLACK, &dpy->color1);
        XParseColor(dpy->display, dpy->colormap, COLOR_WHITE, &dpy->color2);
        XAllocColor(dpy->display, dpy->colormap, &dpy->color1);
        XAllocColor(dpy->display, dpy->colormap, &dpy->color2);

        XSelectInput(dpy->display,dpy->window, 
            KeyPressMask | KeyReleaseMask | ButtonPressMask | ButtonReleaseMask);

}

bool lookup_cell(struct global *g, int x, int y) {
    return (g->cells[(y*GRID_SIZE + x)/8] & (1<<(x%8))) != 0;
}


void do_display(struct global *global, struct display *dpy)
{
    XSetBackground(dpy->display, dpy->gc, dpy->color2.pixel);
    XClearWindow(dpy->display, dpy->window);

    for (int x=0; x<GRID_SIZE; x++)
        for (int y=0; y<GRID_SIZE; y++)
        {
            bool state = get_cell(global, x, y);
            if (state) {
                XSetForeground(dpy->display, dpy->gc, dpy->color1.pixel);
                XFillRectangle(dpy->display, dpy->window, dpy->gc, x*CELL_SIZE, y*CELL_SIZE, CELL_SIZE, CELL_SIZE);        
//              XDrawPoint(dpy->display, dpy->window, dpy->gc, x*CELL_SIZE, y*CELL_SIZE);
            }
        }

    XFlush(dpy->display);
}

void close_display(struct display *dpy)
{
    XDestroyWindow(dpy->display, dpy->window);
    XCloseDisplay(dpy->display);
}

/***********************
    Main program
************************/

void load_life(struct global *g, const char *fname) {
    char *line=NULL;
    size_t len = 0;
    ssize_t nread;
    int x,y;
    FILE *f = fopen(fname, "r");
    if (f==NULL) {
        fprintf(stderr,"Can't open file\n");
        exit(1);
    }
    while ((nread = getline(&line, &len, f)) != -1) {
        if (line[0]=='#')
            continue;
        if (nread<=1)
            continue;
        if (line[0]==13 || line[0]==10)
            continue;
        if (sscanf(line, "%d %d", &x, &y) != 2)
            continue;
        set_cell(g,x+GRID_SIZE/2,y+GRID_SIZE/2,1);
#ifdef CUDA
        // we only need to copy data to over to GPU once
        int index = ((x+GRID_SIZE/2)%GRID_SIZE)*GRID_SIZE+((y+GRID_SIZE/2)%GRID_SIZE);
        g->cellsInt[index] = 1;
        cudaMemcpy(g->gpu_cells, g->cellsInt, GRID_SIZE*GRID_SIZE * sizeof(int), cudaMemcpyHostToDevice);
#endif
    }
    if (line)
        free(line);
    fclose(f);
}

void do_life(struct global *global) {
    bool running=1;
    struct display dpy;
    init_display(&dpy);
    while (running) {
        do_display(global, &dpy);
        usleep(DELAY);
        update(global);

        if (XPending(dpy.display)) {
            XEvent event;
            XNextEvent(dpy.display, &event);
            switch (event.type)
            {
                case ClientMessage:
                    if (event.xclient.data.l[0] == dpy.delete_window)
                        running=0;
                    break;
                case KeyPress:
                case ButtonPress:
                    running=0;
                    break;
                default:
                    break;
            }
        }
    }
    close_display(&dpy);
}

void perf_test(struct global *global) {
    int counter=10000;
    clock_t start = clock();
    clock_t diff;
    int msec;

    printf("Running performance test with %d iterations...\n", counter);
    fflush(stdout);

    while (counter>0) {
        update(global);
        counter--;
    }
    diff = clock() - start;
    msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);
}

int main(int argc, char *argv[]) {
    bool gui=1;
    struct global global;
    init_global(&global);

    #ifdef CUDA
        printf("Starting CUDA version of life....\n");
    #else
        printf("Starting CPU version of life....\n");
    #endif

    int argi;
    for (argi = 1; argi<argc; argi++)
        if (argv[argi][0]=='-' && argv[argi][1]=='i' && argv[argi][2]=='\0')
            gui=0;
        else
            break;

    if (argi==argc-1)
        load_life(&global, argv[argi]);
    else {
        fprintf(stderr,"Syntax: %s [-i] fname.lif\n", argv[0]);
        exit(1);
    }

    if (gui)
        do_life(&global);
    else
        perf_test(&global);
        
    return 0;
}
